#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdint.h>
#include <>
#include <hip/hip_cooperative_groups.h>

#include "ff_wrapper.cu"

template <typename Field_A, typename Field_B>
struct TensorAlgebra
{
    Field_A coeffs[Field_B::EXTENSION_DEGREE][Field_B::EXTENSION_DEGREE];

    __device__ static void add(TensorAlgebra<Field_A, Field_B> *a,
                               TensorAlgebra<Field_A, Field_B> *b,
                               TensorAlgebra<Field_A, Field_B> *result)
    {
        // Works even if result is the same as a or b
        for (int i = 0; i < Field_B::EXTENSION_DEGREE; i++)
        {
            for (int j = 0; j < Field_B::EXTENSION_DEGREE; j++)
            {
                ADD_AA(a->coeffs[i][j], b->coeffs[i][j], result->coeffs[i][j]);
            }
        }
    }

    __device__ static void phi_0_times_phi_1(Field_B *a,
                                             Field_B *b,
                                             TensorAlgebra<Field_A, Field_B> *result)
    {
        for (int i = 0; i < Field_B::EXTENSION_DEGREE; i++)
        {
            for (int j = 0; j < Field_B::EXTENSION_DEGREE; j++)
            {
                MUL_AA(a->coeffs[i], b->coeffs[j], result->coeffs[i][j]);
            }
        }
    }
};

extern "C" __global__ void tensor_algebra_dot_product(Field_B *left, Field_B *right, Field_A *buff, Field_A *result, uint32_t log_len, uint32_t log_n_tasks_per_thread)
{
    // left and right have size 2^log_len
    // buff has size EXT_DEGREE^2 * 2^(log_len - log_n_tasks_per_thread)
    // res has size EXT_DEGREE^2

    namespace cg = cooperative_groups;
    cg::grid_group grid = cg::this_grid();

    int n_total_threads = blockDim.x * gridDim.x;
    int n_reps = ((1 << (log_len - log_n_tasks_per_thread)) + n_total_threads - 1) / n_total_threads;
    for (int rep = 0; rep < n_reps; rep++)
    {
        int idx = threadIdx.x + (blockIdx.x + rep * gridDim.x) * blockDim.x;
        if (idx >= 1 << (log_len - log_n_tasks_per_thread))
        {
            break;
        }
        TensorAlgebra<Field_A, Field_B> sum = {0};
        for (int task = 0; task < 1 << log_n_tasks_per_thread; task++)
        {
            int offset = idx * (1 << log_n_tasks_per_thread) + task;
            Field_B l = left[offset];
            Field_B r = right[offset];
            TensorAlgebra<Field_A, Field_B> res;
            TensorAlgebra<Field_A, Field_B>::phi_0_times_phi_1(&l, &r, &res);
            TensorAlgebra<Field_A, Field_B>::add(&sum, &res, &sum);
        }
        int shift = 0;
        for (int i = 0; i < Field_B::EXTENSION_DEGREE; i++)
        {
            for (int j = 0; j < Field_B::EXTENSION_DEGREE; j++)
            {
                buff[shift + idx] = sum.coeffs[i][j];
                shift += 1 << (log_len - log_n_tasks_per_thread);
            }
        }
    }

    int w = log_len - log_n_tasks_per_thread;
    // Sum
    for (int step = 0; step < w; step++)
    {
        grid.sync();
        int half_size = 1 << (w - step - 1);
        int n_ops = half_size * Field_B::EXTENSION_DEGREE * Field_B::EXTENSION_DEGREE;
        n_reps = (n_ops + n_total_threads - 1) / n_total_threads;
        for (int rep = 0; rep < n_reps; rep++)
        {
            int thread_index = threadIdx.x + (blockIdx.x + rep * gridDim.x) * blockDim.x;
            if (thread_index < n_ops)
            {
                int offset = (thread_index / half_size) << w;
                int m = thread_index % half_size;
                ADD_AA(buff[offset + m], buff[offset + m + half_size], buff[offset + m]);
            }
        }
    }

    grid.sync();
    n_reps = (Field_B::EXTENSION_DEGREE * Field_B::EXTENSION_DEGREE + n_total_threads - 1) / n_total_threads;
    for (int rep = 0; rep < n_reps; rep++)
    {
        int idx = threadIdx.x + (blockIdx.x + rep * gridDim.x) * blockDim.x;
        if (idx < Field_B::EXTENSION_DEGREE * Field_B::EXTENSION_DEGREE)
        {
            result[idx] = buff[idx << (log_len - log_n_tasks_per_thread)];
        }
    }
}