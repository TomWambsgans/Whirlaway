#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <algorithm>

#include "../ff_wrapper.cu"

__device__ int index_transpose(int i, int log_width, int log_len)
{
    int col = i % (1 << log_width);
    int row = i / (1 << log_width);
    return col * (1 << (log_len - log_width)) + row;
}

extern "C" __global__ void reverse_bit_order_for_ntt(Field_A *input, Field_A *output, uint32_t log_len, uint32_t log_expansion_factor, uint32_t log_chunk_size, uint32_t inner_transposition_log_rows)
{
    // 0) Interleave everything with zeros to increase the size of by 1 << log_expansion_factor_u32
    // 1) Bit reverse (everything)
    // 2) transpose (log_expansion_factor, log_len)
    // 3) on each consecutive chunk of size Z^log_chunk_size, transpose (inner_transposition_log_rows, log_chunk_size - inner_transposition_log_rows)

    int total_threads = blockDim.x * gridDim.x;
    const uint32_t n_repetitions = ((1 << log_len) + total_threads - 1) / total_threads;
    int log_expanded_len = log_len + log_expansion_factor;

    for (int rep = 0; rep < n_repetitions; rep++)
    {
        int i = threadIdx.x + (blockIdx.x + gridDim.x * rep) * blockDim.x;
        int log_width = log_expanded_len - log_chunk_size;
        int k = index_transpose(__brev(i * (1 << log_expansion_factor)) >> (32 - log_expanded_len), log_width, log_expanded_len);

        int l = (k >> log_chunk_size) << log_chunk_size;
        int m = k % (1 << log_chunk_size);


        int n = m % (1 << (log_chunk_size - inner_transposition_log_rows));
        int o = m / (1 << (log_chunk_size - inner_transposition_log_rows));
        output[l + (1 << inner_transposition_log_rows) * n + o] = input[i];
    }
}
