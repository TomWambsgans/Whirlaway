#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <algorithm>

#include "../ff_wrapper.cu"

// we need: MAX_NTT_SIZE_AT_BLOCK_LEVEL * (EXT_DEGREE + 1) * 4 bytes <= shared memory
// TODO avoid hardcoding
#if !defined(MAX_NTT_LOG_SIZE_AT_BLOCK_LEVEL)
#define MAX_NTT_LOG_SIZE_AT_BLOCK_LEVEL 1
#endif

extern "C" __global__ void ntt_at_block_level(Field_B *buff, uint32_t log_len, uint32_t log_chunck_size, Field_A *twiddles)
{
    // *twiddles: w^0, w^1, w^2, w^3, ..., w^(log_chunck_size * 2 - 1) where w is a "2 * log_chunck_size" root of unity

    if (log_chunck_size == 0)
    {
        return;
    }

    int threadId = threadIdx.x;
    int n_threads = blockDim.x;

    const int log_n_threads_per_block = __ffs(blockDim.x) - 1;

    const uint32_t n_repetitions = (1 << log_len) / (blockDim.x * gridDim.x * 2);

    __shared__ Field_B cached_buff[1 << MAX_NTT_LOG_SIZE_AT_BLOCK_LEVEL];
    __shared__ Field_A cached_twiddles[1 << (MAX_NTT_LOG_SIZE_AT_BLOCK_LEVEL - 1)]; // TODO use constant memory instead

    if (threadId < (1 << log_chunck_size))
    {
        cached_twiddles[threadId] = twiddles[threadId];
    }

    for (int rep = 0; rep < n_repetitions; rep++)
    {
        int block = blockIdx.x + gridDim.x * rep;

        cached_buff[threadId] = buff[threadId + n_threads * 2 * block];
        cached_buff[threadId + n_threads] = buff[threadId + n_threads * (2 * block + 1)];

        __syncthreads();

        for (int step = 0; step < log_chunck_size; step++)
        {
            int threadIndex = threadIdx.x;
            int fft_index = threadIndex / (1 << (log_chunck_size - 1));
            threadIndex = threadIndex % (1 << (log_chunck_size - 1));

            int inner_fft_size = 1 << step;
            int left_shift = fft_index * (1 << log_chunck_size) + (threadIndex / inner_fft_size);
            int interspace = 1 << (log_chunck_size - step - 1);
            int even_src = left_shift + (threadIndex % inner_fft_size) * 2 * interspace;
            int odd_src = left_shift + ((threadIndex % inner_fft_size) * 2 + 1) * interspace;
            int even_dest = left_shift + (threadIndex % inner_fft_size) * interspace;
            int odd_dest = left_shift + ((threadIndex % inner_fft_size) + inner_fft_size) * interspace;

            Field_B even = cached_buff[even_src];
            Field_B odd = cached_buff[odd_src];

            __syncthreads();

            int i = threadId % inner_fft_size;
            // w^i where w is a "2 * packet_size" root of unity
            Field_A twiddle = cached_twiddles[i * (1 << (log_chunck_size - step - 1))];

            // cached_buff[even_index] = even + first_twiddle * odd
            Field_B temp;
            MUL_BA(odd, twiddle, temp);
            ADD_BB(even, temp, cached_buff[even_dest]);

            // cached_buff[odd_index] = even + second_twiddle * odd
            MUL_BA(odd, twiddle, temp);
            SUB_BB(even, temp, cached_buff[odd_dest]);

            __syncthreads();
        }

        // copy back to global memory
        buff[threadId + blockDim.x * 2 * block] = cached_buff[threadId];
        buff[threadId + blockDim.x * (2 * block + 1)] = cached_buff[threadId + blockDim.x];

        __syncthreads();
    }
}

extern "C" __global__ void apply_twiddles(Field_B *buff, uint32_t full_log_len, uint32_t inner_log_len, uint32_t log_chunck_size, Field_A *twiddles)
{
    int total_threads = blockDim.x * gridDim.x;
    const uint32_t n_repetitions = ((1 << full_log_len) + total_threads - 1) / total_threads;

    for (int rep = 0; rep < n_repetitions; rep++)
    {
        int threadIndex = threadIdx.x + (blockIdx.x + gridDim.x * rep) * blockDim.x;
        int inner_matrix_index = threadIndex / (1 << inner_log_len);

        int inner_idx = threadIndex % (1 << inner_log_len);
        int i = inner_idx % (1 << log_chunck_size);
        int j = inner_idx / (1 << log_chunck_size);
        int ij = i * j;

        Field_A twiddle;
        if (ij < 1 << (inner_log_len - 1))
        {
            twiddle = twiddles[ij];
        }
        else
        {
            twiddle = twiddles[ij - (1 << (inner_log_len - 1))];
            SUB_AA({0}, twiddle, twiddle);
        }

        Field_B src = buff[threadIndex];
        Field_B result;
        MUL_AB(twiddle, src, result);
        buff[threadIndex] = result;
    }
}