#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <algorithm>

#include "../ff_wrapper.cu"

// we need: MAX_NTT_SIZE_AT_BLOCK_LEVEL * (EXT_DEGREE + 1) * 4 bytes <= shared memory
// TODO avoid hardcoding
#if !defined(MAX_NTT_LOG_SIZE_AT_BLOCK_LEVEL)
#define MAX_NTT_LOG_SIZE_AT_BLOCK_LEVEL 0
#endif

extern "C" __global__ void ntt_at_block_level(Field_B *buff, uint32_t log_len, uint32_t log_chunck_size, Field_A *twiddles)
{
    // *twiddles: w^0, w^1, w^2, w^3, ..., w^(log_chunck_size * 2 - 1) where w is a "2 * log_chunck_size" root of unity

    if (log_chunck_size == 0)
    {
        return;
    }

    int threadId = threadIdx.x;
    int n_threads = blockDim.x;

    const int log_n_threads_per_block = __ffs(blockDim.x) - 1;

    const uint32_t n_repetitions = (1 << log_len) / (blockDim.x * gridDim.x * 2);

    __shared__ Field_B cached_buff[1 << MAX_NTT_LOG_SIZE_AT_BLOCK_LEVEL];
    __shared__ Field_A cached_twiddles[1 << MAX_NTT_LOG_SIZE_AT_BLOCK_LEVEL]; // TODO use constant memory instead

    cached_twiddles[threadId] = twiddles[n_threads * 2 - 1 + threadId];
    cached_twiddles[threadId + n_threads] = twiddles[n_threads * 3 - 1 + threadId];

    for (int rep = 0; rep < n_repetitions; rep++)
    {
        int block = blockIdx.x + gridDim.x * rep;

        cached_buff[threadId] = buff[threadId + n_threads * 2 * block];
        cached_buff[threadId + n_threads] = buff[threadId + n_threads * (2 * block + 1)];

        __syncthreads();

        for (int step = 0; step < log_chunck_size; step++)
        {
            int threadIndex = threadIdx.x;
            int fft_index = threadIndex / (1 << (log_chunck_size - 1));
            threadIndex = threadIndex % (1 << (log_chunck_size - 1));

            int inner_fft_size = 1 << step;
            int left_shift = fft_index * (1 << log_chunck_size) + (threadIndex / inner_fft_size);
            int interspace = 1 << (log_chunck_size - step - 1);
            int even_src = left_shift + (threadIndex % inner_fft_size) * 2 * interspace;
            int odd_src = left_shift + ((threadIndex % inner_fft_size) * 2 + 1) * interspace;
            int even_dest = left_shift + (threadIndex % inner_fft_size) * interspace;
            int odd_dest = left_shift + ((threadIndex % inner_fft_size) + inner_fft_size) * interspace;

            Field_B even = cached_buff[even_src];
            Field_B odd = cached_buff[odd_src];

            __syncthreads();

            int i = threadId % inner_fft_size;
            // w^i where w is a "2 * packet_size" root of unity
            Field_A first_twiddle = cached_twiddles[i * blockDim.x / inner_fft_size];
            // w^(i + packet_size) where w is a "2 * packet_size" root of unity
            Field_A second_twiddle = cached_twiddles[(i + inner_fft_size) * blockDim.x / inner_fft_size];

            // cached_buff[even_index] = even + first_twiddle * odd
            Field_B temp;
            MUL_BA(odd, first_twiddle, temp);
            ADD_BB(even, temp, cached_buff[even_dest]);

            // cached_buff[odd_index] = even + second_twiddle * odd
            MUL_BA(odd, second_twiddle, temp);
            ADD_BB(even, temp, cached_buff[odd_dest]);

            __syncthreads();
        }

        // copy back to global memory
        buff[threadId + blockDim.x * 2 * block] = cached_buff[threadId];
        buff[threadId + blockDim.x * (2 * block + 1)] = cached_buff[threadId + blockDim.x];

        __syncthreads();
    }
}

extern "C" __global__ void apply_twiddles(Field_B *buff, uint32_t full_log_len, uint32_t inner_log_len, uint32_t log_chunck_size, Field_A *twiddles)
{
    int total_threads = blockDim.x * gridDim.x;
    const uint32_t n_repetitions = ((1 << full_log_len) + total_threads - 1) / total_threads;

    for (int rep = 0; rep < n_repetitions; rep++)
    {
        int threadIndex = threadIdx.x + (blockIdx.x + gridDim.x * rep) * blockDim.x;
        int inner_matrix_index = threadIndex / (1 << inner_log_len);

        int inner_idx = threadIndex % (1 << inner_log_len);
        int i = inner_idx % (1 << log_chunck_size);
        int j = inner_idx / (1 << log_chunck_size);
        Field_A twiddle = twiddles[(1 << inner_log_len) - 1 + i * j];

        Field_B src = buff[threadIndex];
        Field_B result;
        MUL_AB(twiddle, src, result);
        buff[threadIndex] = result;
    }
}