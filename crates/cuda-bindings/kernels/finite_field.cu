
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#ifdef USE_NOINLINE
#define MAYBE_NOINLINE __noinline__
#else
#define MAYBE_NOINLINE
#endif

// Koala bear field
#define MONTY_PRIME 0x7f000001
#define MONTY_BITS 32
#define MONTY_MASK ((1ULL << MONTY_BITS) - 1U)
#define MONTY_MU 0x81000001

#define EXT_DEGREE 8
#define W 100663290U // montgomery representation of 3, X^8 - 3 is irreducible

__device__ constexpr uint32_t to_monty(uint32_t x)
{
    return (uint32_t)(((uint64_t)x << MONTY_BITS) % MONTY_PRIME);
}

__device__ uint32_t monty_reduce(uint64_t x)
{
    uint64_t t = x * MONTY_MU & MONTY_MASK;
    uint64_t u = t * MONTY_PRIME;

    uint64_t x_sub_u = x - u;
    bool over = x < u;
    uint32_t x_sub_u_hi = (x_sub_u >> MONTY_BITS);
    uint32_t corr = over ? MONTY_PRIME : 0;
    return x_sub_u_hi + corr;
}

__device__ uint32_t monty_field_add(const uint32_t a, const uint32_t b)
{
    uint32_t sum = a + b;
    if (sum >= MONTY_PRIME)
    {
        sum -= MONTY_PRIME;
    }
    return sum;
}

__device__ uint32_t monty_field_mul(const uint32_t a, const uint32_t b)
{
    uint64_t long_prod = (uint64_t)a * (uint64_t)b;
    return monty_reduce(long_prod);
}

__device__ uint32_t monty_field_sub(const uint32_t a, const uint32_t b)
{
    uint32_t diff = a - b;
    bool over = a < b; // Detect underflow
    uint32_t corr = over ? MONTY_PRIME : 0;
    return diff + corr;
}

typedef struct
{
    uint32_t coeffs[EXT_DEGREE]; // Polynomial coefficients
} ExtField;

__device__ void print_ext_field(const ExtField *a)
{
    for (int i = 0; i < EXT_DEGREE; i++)
    {
        printf("%u ", a->coeffs[i]);
    }
    printf("\n");
}

// Add two extension field elements
__device__ MAYBE_NOINLINE void ext_field_add(const ExtField *a, const ExtField *b, ExtField *result)
{
    // Works even if result is the same as a or b
    for (int i = 0; i < EXT_DEGREE; i++)
    {
        result->coeffs[i] = monty_field_add(a->coeffs[i], b->coeffs[i]);
    }
}

// Subtract two extension field elements
__device__ MAYBE_NOINLINE void ext_field_sub(const ExtField *a, const ExtField *b, ExtField *result)
{
    // Works even if result is the same as a or b
    for (int i = 0; i < EXT_DEGREE; i++)
    {
        result->coeffs[i] = monty_field_sub(a->coeffs[i], b->coeffs[i]);
    }
}

__device__ MAYBE_NOINLINE void mul_prime_and_ext_field(const ExtField *a, uint32_t b, ExtField *result)
{
    // Works even if result is the same as a
    for (int i = 0; i < EXT_DEGREE; i++)
    {
        result->coeffs[i] = monty_field_mul(a->coeffs[i], b);
    }
}

__device__ MAYBE_NOINLINE void add_prime_and_ext_field(const ExtField *a, uint32_t b, ExtField *result)
{
    // TODO this would be more efficient in place (to avoid the copy loop)

    result->coeffs[0] = monty_field_add(a->coeffs[0], b);
    for (int i = 1; i < EXT_DEGREE; i++)
    {
        result->coeffs[i] = a->coeffs[i];
    }
}

// TODO Karatsuba ?
__device__ MAYBE_NOINLINE void ext_field_mul(const ExtField *a, const ExtField *b, ExtField *result)
{
    // Does not work if result is the same as a or b
    for (int i = 0; i < EXT_DEGREE; i++)
    {
        result->coeffs[i] = 0;
    }

    // Schoolbook multiplication
    for (int i = 0; i < EXT_DEGREE; i++)
    {
        for (int j = 0; j < EXT_DEGREE; j++)
        {

            uint32_t prod = monty_field_mul(a->coeffs[i], b->coeffs[j]);

            if (i + j < EXT_DEGREE)
            {
                uint32_t temp = monty_field_add(result->coeffs[i + j], prod);
                result->coeffs[i + j] = temp;
            }
            else
            {
                uint32_t temp = monty_field_mul(prod, W);
                result->coeffs[i + j - EXT_DEGREE] = monty_field_add(result->coeffs[i + j - EXT_DEGREE], temp);
            }
        }
    }
}