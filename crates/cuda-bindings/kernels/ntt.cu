#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_cooperative_groups.h>

// Koala bear field
#define MONTY_PRIME 0x7f000001
#define MONTY_BITS 32
#define MONTY_MASK ((1ULL << MONTY_BITS) - 1U)
#define MONTY_MU 0x81000001

#define EXT_DEGREE 8
#define W 100663290U // montgomery representation of 3, X^8 - 3 is irreducible

// we need: thread_per_block * 2 * (EXT_DEGREE + 1) * 4 bytes <= shared memory
// TODO avoid hardcoding
#define LOG_THREAD_PER_BLOCK 8
#define THREAD_PER_BLOCK (1 << LOG_THREAD_PER_BLOCK)

__device__ uint32_t monty_reduce(uint64_t x)
{
    uint64_t t = x * MONTY_MU & MONTY_MASK;
    uint64_t u = t * MONTY_PRIME;

    uint64_t x_sub_u = x - u;
    bool over = x < u;
    uint32_t x_sub_u_hi = (x_sub_u >> MONTY_BITS);
    uint32_t corr = over ? MONTY_PRIME : 0;
    return x_sub_u_hi + corr;
}

// CUDA kernel for field addition
__device__ void monty_field_add(const uint32_t a, const uint32_t b, uint32_t *result)
{
    uint32_t sum = a + b;
    if (sum >= MONTY_PRIME)
    {
        sum -= MONTY_PRIME;
    }
    *result = sum;
}

// CUDA kernel for field multiplication
__device__ void monty_field_mul(const uint32_t a, const uint32_t b, uint32_t *result)
{
    uint64_t long_prod = (uint64_t)a * (uint64_t)b;
    *result = monty_reduce(long_prod);
}

// CUDA kernel for field subtraction
__device__ void monty_field_sub(const uint32_t a, const uint32_t b, uint32_t *result)
{
    uint32_t diff = a - b;
    bool over = a < b; // Detect underflow
    uint32_t corr = over ? MONTY_PRIME : 0;
    *result = diff + corr;
}

// Extension field implementation
typedef struct
{
    uint32_t coeffs[EXT_DEGREE]; // Polynomial coefficients
} ExtField;

__device__ void print_ext_field(const ExtField *a)
{
    for (int i = 0; i < EXT_DEGREE; i++)
    {
        printf("%u ", a->coeffs[i]);
    }
    printf("\n");
}

// Add two extension field elements
__device__ void ext_field_add(const ExtField *a, const ExtField *b, ExtField *result)
{
    for (int i = 0; i < EXT_DEGREE; i++)
    {
        monty_field_add(a->coeffs[i], b->coeffs[i], &result->coeffs[i]);
    }
}

// Subtract two extension field elements
__device__ void ext_field_sub(const ExtField *a, const ExtField *b, ExtField *result)
{
    for (int i = 0; i < EXT_DEGREE; i++)
    {
        monty_field_sub(a->coeffs[i], b->coeffs[i], &result->coeffs[i]);
    }
}

__device__ void mul_prime_by_ext_field(const ExtField *a, uint32_t b, ExtField *result)
{
    for (int i = 0; i < EXT_DEGREE; i++)
    {
        monty_field_mul(a->coeffs[i], b, &result->coeffs[i]);
    }
}

// TODO Karatsuba ?
__device__ void ext_field_mul(const ExtField *a, const ExtField *b, ExtField *result)
{
    for (int i = 0; i < EXT_DEGREE; i++)
    {
        result->coeffs[i] = 0;
    }

    // Schoolbook multiplication
    for (int i = 0; i < EXT_DEGREE; i++)
    {
        for (int j = 0; j < EXT_DEGREE; j++)
        {
            uint32_t prod;
            monty_field_mul(a->coeffs[i], b->coeffs[j], &prod);

            if (i + j < EXT_DEGREE)
            {
                uint32_t temp;
                monty_field_add(result->coeffs[i + j], prod, &temp);
                result->coeffs[i + j] = temp;
            }
            else
            {
                uint32_t temp;
                monty_field_mul(prod, W, &temp);
                monty_field_add(result->coeffs[i + j - EXT_DEGREE], temp, &temp);
                result->coeffs[i + j - EXT_DEGREE] = temp;
            }
        }
    }
}

__device__ void ntt_at_block_level(ExtField *buff, const int block, const uint32_t *twiddles)
{
    // the initial steps of the NTT are done at block level, to make use of shared memory
    // *buff constains THREAD_PER_BLOCK * 2 ExtField elements
    // *twiddles: w^0, w^1, w^2, w^3, ..., w^(THREAD_PER_BLOCK * 2 - 1) where w is a "2 * THREAD_PER_BLOCK" root of unity
    // block is not necessarily blockIdx.x

    const int threadId = threadIdx.x;

    __shared__ ExtField cached_buff[THREAD_PER_BLOCK * 2];

    cached_buff[threadId] = buff[threadId + THREAD_PER_BLOCK * 2 * block];
    cached_buff[threadId + THREAD_PER_BLOCK] = buff[threadId + THREAD_PER_BLOCK * (2 * block + 1)];

    __shared__ uint32_t cached_twiddles[THREAD_PER_BLOCK * 2];

    cached_twiddles[threadId] = twiddles[threadId];
    cached_twiddles[threadId + THREAD_PER_BLOCK] = twiddles[threadId + THREAD_PER_BLOCK];

    __syncthreads();

    // step 0

    ExtField even = cached_buff[threadId * 2];
    ExtField odd = cached_buff[threadId * 2 + 1];

    ext_field_add(&even, &odd, &cached_buff[threadId * 2]);
    ext_field_sub(&even, &odd, &cached_buff[threadId * 2 + 1]);

    for (int step = 1; step <= LOG_THREAD_PER_BLOCK; step++)
    {
        int packet_size = 1 << step;
        int even_index = threadId + (threadId / packet_size) * packet_size;
        int odd_index = even_index + packet_size;

        ExtField even = cached_buff[even_index];
        ExtField odd = cached_buff[odd_index];

        int i = threadId % packet_size;
        // w^i where w is a "2 * packet_size" root of unity
        uint32_t first_twiddle = cached_twiddles[i * THREAD_PER_BLOCK / packet_size];
        // w^(i + packet_size) where w is a "2 * packet_size" root of unity
        uint32_t second_twiddle = cached_twiddles[(i + packet_size) * THREAD_PER_BLOCK / packet_size];

        // cached_buff[even_index] = even + first_twiddle * odd
        mul_prime_by_ext_field(&odd, first_twiddle, &cached_buff[even_index]);
        ext_field_add(&even, &cached_buff[even_index], &cached_buff[even_index]);

        // cached_buff[odd_index] = even + second_twiddle * odd
        mul_prime_by_ext_field(&odd, second_twiddle, &cached_buff[odd_index]);
        ext_field_add(&even, &cached_buff[odd_index], &cached_buff[odd_index]);

        __syncthreads();
    }

    // copy back to global memory
    buff[threadId + THREAD_PER_BLOCK * 2 * block] = cached_buff[threadId];
    buff[threadId + THREAD_PER_BLOCK * (2 * block + 1)] = cached_buff[threadId + THREAD_PER_BLOCK];
}

__device__ void reverse_bit_order(ExtField *data, int block, int bits)
{
    int idx = (block * blockDim.x + threadIdx.x) % (1 << bits);
    int rev_idx = __brev(idx) >> (32 - bits);

    // Only process when idx < rev_idx to avoid swapping twice
    if (idx < rev_idx)
    {
        ExtField temp = data[idx];
        data[idx] = data[rev_idx];
        data[rev_idx] = temp;
    }
}

__device__ void batch_reverse_bit_order(ExtField *data, int block, int bits)
{
    int idx = block * blockDim.x + threadIdx.x;
    int len = (1 << bits);
    reverse_bit_order(&data[(idx / len) * len], block, bits);
}

// TODO use only one buffer, but I don't know how to fill it "partially" with cudarc, since the crate asserts dest size = src size when copying data
extern "C" __global__ void ntt(ExtField *input, ExtField *buff, ExtField *result, const uint32_t log_len, const uint32_t log_extension_factor, const uint32_t *twiddles)
{
    // twiddles = 1
    // followed by w^0, w^1 where w is a 2-root of unity
    // followed by w^0, w^1, w^2, w^3 where w is a 4-root of unity
    // followed by w^0, w^1, w^2, w^3, w^4, w^5, w^6, w^7 where w is a 8-root of unity
    // ...
    // input has size 1 << log_len (the coefs of the polynomial we want to NTT)
    // buff and result both have size 1 << (log_len + log_extension_factor)

    namespace cg = cooperative_groups;
    cg::grid_group grid = cg::this_grid();

    // we should have THREAD_PER_BLOCK * NUM_BLOCKS * n_repetitions * 2 = 1 << (log_len + log_extension_factor)
    // WARNING: We assume the number of blocks is a power of 2
    const uint32_t n_repetitions = (1 << (log_len + log_extension_factor)) / (THREAD_PER_BLOCK * gridDim.x * 2);

    // int threadIndex = threadIdx.x + blockIdx.x * blockDim.x;

    const int len = 1 << log_len;
    const int expansion_factor = 1 << log_extension_factor;

    // 1) Expand input several times to fill result, multiplying by the appropriate twiddle factors
    for (int rep = 0; rep < n_repetitions * 2; rep++)
    {
        int threadIndex = threadIdx.x + (blockIdx.x + gridDim.x * rep) * THREAD_PER_BLOCK;

        if (threadIndex < len)
        {
            buff[threadIndex] = input[threadIndex];
        }
        else
        {
            uint32_t twidle = twiddles[(1 << (log_len + log_extension_factor)) - 1 + (threadIndex % len) * (threadIndex / len)];
            mul_prime_by_ext_field(&input[threadIndex % len], twidle, &buff[threadIndex]);
        }
    }

    grid.sync();

    // 2) Bit reverse order

    for (int rep = 0; rep < n_repetitions * 2; rep++)
    {
        batch_reverse_bit_order(buff, blockIdx.x + gridDim.x * rep, log_len);
    }

    grid.sync();

    // 3) Do the NTT at block level

    for (int rep = 0; rep < n_repetitions; rep++)
    {
        ntt_at_block_level(buff, blockIdx.x + gridDim.x * rep, &twiddles[THREAD_PER_BLOCK * 2 - 1]);
    }

    // 4) Finish the NTT

    for (int step = LOG_THREAD_PER_BLOCK + 1; step < log_len; step++)
    {
        grid.sync();

        // we group together pairs which each side contains 1 << step elements

        for (int rep = 0; rep < n_repetitions; rep++)
        {
            int threadIndex = threadIdx.x + (blockIdx.x + gridDim.x * rep) * THREAD_PER_BLOCK;

            int packet_size = 1 << step;
            int even_index = threadIndex + (threadIndex / packet_size) * packet_size;
            int odd_index = even_index + packet_size;

            ExtField even = buff[even_index];
            ExtField odd = buff[odd_index];

            int i = threadIndex % packet_size;
            // w^i where w is a "2 * packet_size" root of unity
            uint32_t first_twiddle = twiddles[packet_size * 2 - 1 + i];
            // w^(i + packet_size) where w is a "2 * packet_size" root of unity
            uint32_t second_twiddle = twiddles[packet_size * 2 - 1 + i + packet_size];

            // result[even_index] = even + first_twiddle * odd
            mul_prime_by_ext_field(&odd, first_twiddle, &buff[even_index]);
            ext_field_add(&even, &buff[even_index], &buff[even_index]);

            // result[odd_index] = even + second_twiddle * odd
            mul_prime_by_ext_field(&odd, second_twiddle, &buff[odd_index]);
            ext_field_add(&even, &buff[odd_index], &buff[odd_index]);
        }
    }

    grid.sync();

    // 5) Transpose buff to result

    for (int rep = 0; rep < n_repetitions * 2; rep++)
    {
        int threadIndex = threadIdx.x + (blockIdx.x + gridDim.x * rep) * THREAD_PER_BLOCK;

        result[threadIndex] = buff[(threadIndex % expansion_factor) * len + (threadIndex / expansion_factor)];
    }
}
